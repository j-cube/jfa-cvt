/**
 * Copyright (c) 2014, Bo Zhou<Bo.Schwarzstein@gmail.com> and J CUBE Inc. Tokyo, Japan
 * All rights reserved.

 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 1. Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 * 3. All advertising materials mentioning features or use of this software
 *    must display the following acknowledgement:
 *    This product includes software developed by the <organization>.
 * 4. Neither the name of the <organization> nor the
 *    names of its contributors may be used to endorse or promote products
 *    derived from this software without specific prior written permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY COPYRIGHT HOLDER AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL COPYRIGHT HOLDER BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <ctime>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <iostream>
#include <iterator>
#include <vector>

__global__ void Kernel( int SizeX , int SizeY , const float2 * SiteArray , const int * Ping , int * Pong , int k , int * Mutex )
{
    //
    const int CellX = threadIdx.x + blockIdx.x * blockDim.x ;
    const int CellY = threadIdx.y + blockIdx.y * blockDim.y ;

    const int CellIdx = CellX + CellY * SizeX ;
    const int Seed = Ping[CellIdx] ;
    if ( Seed < 0 )
    {
        return ;
    }

    //
    const int2 OffsetArray[8] = { { - 1 , - 1 } ,
                                  {   0 , - 1 } ,
                                  {   1 , - 1 } ,
                                  { - 1 ,   0 } ,
                                  {   1 ,   0 } ,
                                  { - 1 ,   1 } ,
                                  {   0 ,   1 } ,
                                  {   1 ,   1 } } ;

    for ( int i = 0 ; i < 8 ; ++ i )
    {
        const int FillCellX = CellX + k * OffsetArray[i].x ;
        const int FillCellY = CellY + k * OffsetArray[i].y ; 
        if ( FillCellX >= 0 && FillCellX < SizeX && FillCellY >= 0 && FillCellY < SizeY )
        {
            //
            const int FillCellIdx = FillCellX + FillCellY * SizeX ;

            // Lock
            //
            while ( atomicCAS( Mutex , - 1 , FillCellIdx ) == FillCellIdx )
            {
            }

            const int FillSeed = Pong[FillCellIdx] ;

            if ( FillSeed < 0 )
            {
                Pong[FillCellIdx] = Seed ;
            }
            else
            {
                float2 P = make_float2( FillCellX + 0.5f , FillCellY + 0.5f ) ;

                float2 A = SiteArray[Seed] ;
                float2 PA = make_float2( A.x - P.x , A.y - P.y ) ;
                float PALength = PA.x * PA.x + PA.y * PA.y ;

                const float2 B = SiteArray[FillSeed] ;
                float2 PB = make_float2( B.x - P.x , B.y - P.y ) ;
                float PBLength = PB.x * PB.x + PB.y * PB.y ;

                if ( PALength < PBLength )
                {
                    Pong[FillCellIdx] = Seed ;
                }
            }

            // Release
            //
            atomicExch( Mutex , - 1 ) ;
        }
    }
}

int main( int Argc , char * Argv[] )
{
    -- Argc , ++ Argv ;
    if ( Argc != 3 )
    {
        return EXIT_FAILURE ;
    }

    //
    int NumSites = atoi( Argv[0] ) ;
    int Size     = atoi( Argv[1] ) ;

    //
    int NumCudaDevice = 0 ;
    hipGetDeviceCount( & NumCudaDevice ) ;
    if ( ! NumCudaDevice )
    {
        return EXIT_FAILURE ;
    }

    //
    //
    std::vector< float2 > SiteVec ;
    std::vector< int >    SeedVec( Size * Size , - 1 ) ;
    std::vector< uchar3 > RandomColorVec ;
    for ( int i = 0 ; i < NumSites ; ++ i )
    {
        float X = static_cast< float >( rand() ) / RAND_MAX * Size ;
        float Y = static_cast< float >( rand() ) / RAND_MAX * Size ;
        int CellX = static_cast< int >( floorf( X ) ) ;
        int CellY = static_cast< int >( floorf( Y ) ) ;

        SiteVec.push_back( make_float2( CellX + 0.5f , CellY + 0.5f ) ) ;
        SeedVec[CellX + CellY * Size] = i ;

        RandomColorVec.push_back( make_uchar3( static_cast< unsigned char >( static_cast< float >( rand() ) / RAND_MAX * 255.0f ) ,
                                               static_cast< unsigned char >( static_cast< float >( rand() ) / RAND_MAX * 255.0f ) ,
                                               static_cast< unsigned char >( static_cast< float >( rand() ) / RAND_MAX * 255.0f ) ) ) ;
    }

    //
    size_t SiteSize = NumSites * sizeof( float2 ) ;

    float2 * SiteArray = NULL ;
    hipMalloc( & SiteArray , SiteSize ) ;
    hipMemcpy( SiteArray , & SiteVec[0] , SiteSize , hipMemcpyHostToDevice ) ;

    //
    size_t BufferSize = Size * Size * sizeof( int ) ;

    int * Ping = NULL , * Pong = NULL ;
    hipMalloc( & Ping , BufferSize ) , hipMemcpy( Ping , & SeedVec[0] , BufferSize , hipMemcpyHostToDevice ) ;
    hipMalloc( & Pong , BufferSize ) , hipMemcpy( Pong , Ping , BufferSize , hipMemcpyDeviceToDevice ) ;

    //
    int * Mutex = NULL ;
    hipMalloc( & Mutex , sizeof( int ) ) , hipMemset( Mutex , - 1 , sizeof( int ) ) ;

    //
    //
    hipDeviceProp_t CudaDeviceProperty ;
    hipGetDeviceProperties( & CudaDeviceProperty , 0 ) ;

    dim3 BlockDim( CudaDeviceProperty.warpSize , CudaDeviceProperty.warpSize ) ;
    dim3 GridDim( ( Size + BlockDim.x - 1 ) / BlockDim.x ,
                  ( Size + BlockDim.y - 1 ) / BlockDim.y ) ;

    for ( int k = Size / 2 ; k > 0 ; k = k >> 1 )
    {
        Kernel<<< GridDim , BlockDim >>>( Size , Size , SiteArray , Ping , Pong , k , Mutex ) ;
        hipDeviceSynchronize() ;

        hipMemcpy( Ping , Pong , BufferSize , hipMemcpyDeviceToDevice ) ;
        std::swap( Ping , Pong ) ;
    }
    hipMemcpy( & SeedVec[0] , Pong , BufferSize , hipMemcpyDeviceToHost ) ;

    //
    hipFree( SiteArray ) ;
    hipFree( Ping ) ;
    hipFree( Pong ) ;
    hipFree( Mutex ) ;

    //
    //
    FILE * Output = fopen( Argv[2] , "wb" ) ;
    fprintf( Output , "P6\n%d %d\n255\n" , Size , Size ) ;

    std::vector< uchar3 > Pixels( Size * Size ) ;
    for ( int y = 0 ; y < Size ; ++ y )
    {
        for ( int x = 0 ; x < Size ; ++ x )
        {
            const int Seed = SeedVec[x + y * Size] ;
            if ( Seed != - 1 )
            {
                Pixels[x + y * Size] = RandomColorVec[Seed] ;
            }
        }
    }

    for( std::vector< float2 >::const_iterator itr = SiteVec.begin() ; itr != SiteVec.end() ; ++ itr )
    {
        const int x = static_cast< int >( floorf( itr->x ) ) ;
        const int y = static_cast< int >( floorf( itr->y ) ) ;
        Pixels[x + y * Size] = make_uchar3( 255 , 0 , 0 ) ;
    }

    fwrite( & Pixels[0].x , 3 , Pixels.size() , Output ) ;
    fclose( Output ) ;

    return EXIT_SUCCESS ;
}
